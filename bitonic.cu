#include "hip/hip_runtime.h"

__global__  void bitonic_sort_step(float *dev_values, int j, int k)
{
	unsigned int i, ixj; /* Sorting partners: i and ixj */
	i = threadIdx.x + blockDim.x * blockIdx.x;
	ixj = i^j;

	/* The threads with the lowest ids sort the array. */
	if ((ixj) > i) 
	{
		if ((i&k) == 0) 
		{
			/* Sort ascending */
			if (dev_values[i]>dev_values[ixj]) 
			{
				/* exchange(i,ixj); */
				float temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
		if ((i&k) != 0) 
		{
			/* Sort descending */
			if (dev_values[i]<dev_values[ixj]) 
			{
				/* exchange(i,ixj); */
				float temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
	}
}

void bitonic_sort(float *values, int NUM_VALS, int BLOCKS, int THREADS)
{
	float *dev_values;
	size_t size = NUM_VALS * sizeof(float);

	// Allocate sapce for device copies of values
	hipMalloc((void**) &dev_values, size);	
	hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

	// ������� ���������� ���������� ����
	dim3 blocks(BLOCKS, 1);    /* Number of blocks   */
	dim3 threads(THREADS, 1);  /* Number of threads  */

	int j, k;
	/* Major step */
	for (k = 2; k <= NUM_VALS; k <<= 1)
	{
		/* Minor step */
		for (j = k >> 1; j > 0; j >>= 1) 
		{
			// ������ ���� �� ����������
			bitonic_sort_step <<<blocks, threads>>>(dev_values, j, k);
		}
	}
	hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
	hipFree(dev_values);
}
