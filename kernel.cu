#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "iostream"

#include "bitonic.h"

#define THREADS 128
#define BLOCKS 32768
#define NUM_VALS THREADS*BLOCKS

using namespace std;

void array_fill(float *arr, int length)
{
	srand(time(NULL));
	for (int i = 0; i < length; ++i) 
	{
		arr[i] = (float) rand() / (float) RAND_MAX;
	}
}

void Print(float *arr)
{
	for (int i = 0; i < 1000000; i+=1000)
	{
		printf("%f ", arr[i]);
	}
}

int main(void)
{
	float *values = (float*) malloc(NUM_VALS * sizeof(float));	
	array_fill(values, NUM_VALS);

	clock_t start = clock();
	bitonic_sort(values, NUM_VALS, BLOCKS, THREADS);

	cout << "Time: " << ((double) (clock() - start)) / CLOCKS_PER_SEC << " seconds." << endl;
	//Print(values);
}